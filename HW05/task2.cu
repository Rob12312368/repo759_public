#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
__global__ void compute(int* dA, int myrand){

    dA[blockIdx.x * blockDim.x + threadIdx.x] = myrand * threadIdx.x + blockIdx.x;
}
int main(int argc, char* argv []){
    

    int* dA;
    int hA[16];

    std::random_device rd;
    std::mt19937 gen(rd());

    std::uniform_real_distribution<> distrib(5.0, 10.0);

    hipMalloc((void**)&dA, sizeof(int) * 16);
    hipMemset(dA, 0, sizeof(int) * 16);


    compute<<<2, 8>>>(dA, distrib(gen));

    hipMemcpy(hA, dA, sizeof(int) * 16, hipMemcpyDeviceToHost);
    for(int i = 0; i < 16; i++)
        printf(" %d", hA[i]);

    hipFree(dA);
    return 0;
}