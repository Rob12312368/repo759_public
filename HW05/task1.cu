#include <hip/hip_runtime.h>
#include <iostream>



__global__ void factorial(){
    int start = 1;
    for(int i = 0; i < threadIdx.x ; i++){
        start *= (i+2);
    }
    printf("%d!=%d\n", threadIdx.x+1, start);
}

int main(){

    factorial<<<1, 8>>>();
    hipDeviceSynchronize();

}
