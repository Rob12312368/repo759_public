#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include "vscale.cuh"

int main(int argc, char* argv []){
    unsigned int n = std::stoi(argv[1]);

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    

    float* ha = new float[n];
    float* hb = new float[n];
    float* da;
    float* db;
    std::random_device rd;
    std::mt19937 gen(rd());

    std::uniform_real_distribution<float> distrib_a(-10.0, 10.0);
    std::uniform_real_distribution<float> distrib_b(0.0, 1.0);

    for(unsigned int i = 0; i < n ; i++){
        ha[i] = distrib_a(gen);
        hb[i] = distrib_b(gen);
    }

    hipMalloc((void**)&da, sizeof(float) * n);
    hipMalloc((void**)&db, sizeof(float) * n);
    hipMemcpy(da, ha, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, sizeof(float) * n, hipMemcpyHostToDevice);

    hipEventRecord(startEvent, 0);
    vscale<<<1, 16>>>(da, db, n);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);

    hipMemcpy(hb, db, sizeof(float) * n, hipMemcpyDeviceToHost);
    std::cout << elapsedTime << "\n" << hb[0] << "\n" << hb[n-1];

    

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipFree(da);
    hipFree(db);
    delete[] ha;
    delete[] hb;


}